#include "hip/hip_runtime.h"
#include "add.cuh"

namespace chatty {
namespace cuda {

Status add(const Tensor& x, const Tensor& y, Tensor& out) {
    DType dtype = x.dtype();
    if(dtype != y.dtype()) {
        // TODO: log
        return StatusCode::CHATTY_STATUS_FAILURE;
    }

    // dim3 grid((x.shape().num_elements() + 255) / 256);
}

Status add_fp16(const __half* x, const __half* y, __half* out,  int m, int n);
Status add_fp32(const float* x, const float* y, float* out, int m, int n);
Status add_int8(const int8_t* x, const int8_t* y, int8_t* out, int m, int n);
Status add_int16(const int16_t* x, const int16_t* y, int16_t* out, int m, int n);
Status add_int32(const int32_t* x, const int32_t* y, int32_t* out, int m, int n);

__global__
void kernel_add_fp16(const __half* x, const __half* y, __half* out, int m, int n);
__global__
void kernel_add_fp32(const float* x, const float* y, float* out, int m, int n);
__global__
void kernel_add_int8(const int8_t* x, const int8_t* y, int8_t* out, int m, int n);
__global__
void kernel_add_int16(const int16_t* x, const int16_t* y, int16_t* out, int m, int n);
__global__
void kernel_add_int32(const int32_t* x, const int32_t* y, int32_t* out, int m, int n);

} // namespace cuda
} // namespace chatty