#include "hip/hip_runtime.h"
#include "add.cuh"

namespace chatty {
namespace cuda {

Status add(hipblasHandle_t handle, float alpha=1.0f, const Tensor& x, Tensor& y) {
    DType dtype = x.dtype();
    Shape shape = x.shape();
    if(dtype != y.dtype() || shape != y.shape()) {
        // TODO: log
        return StatusCode::CHATTY_STATUS_FAILURE;
    }
    switch(dtype) {
    case DType::Float16:
        return add_fp16(
            handle, alpha, 
            static_cast<const __half*>(x.data()), 
            static_cast<const __half*>(y.data()),
            shape.num_elements()
        );
    }
    case DType::Float32:
        return add_fp32(
            handle, alpha,
            static_cast<const float*>(x.data()), 
            static_cast<const float*>(y.data()),
        );
}

Status add_fp16(hipblasHandle_t handle, float alpha=1.0f, const __half* x, __half* y, int m, int n) {
    // hipblasStatus_t status = hipblasGemmEx(
    //     handle,          // cuBLAS 上下文句柄
    //     HIPBLAS_OP_N,       // A 矩阵转置标志（N/T/C）
    //     HIPBLAS_OP_N,       // B 矩阵转置标志（N/T/C）
    //     m, n, k,            // 矩阵维度：C = [m×n], A = [m×k], B = [k×n]
    //     &alpha,              // 标量系数 α
    //     x,                  // A 矩阵指针
    //     HIP_R_16F,           // A 数据类型（如 HIP_R_16F）
    //     m,                        // A 的主维度（列主序下为行数）
    //     y,                  // B 矩阵指针
    //     HIP_R_16F,           // B 数据类型
    //     k,                        // B 的主维度
    //     const void *beta,               // 标量系数 β
    //     void *C,                        // C 矩阵指针（输出）
    //     hipDataType Ctype,           // C 数据类型
    //     int ldc,                        // C 的主维度
    //     hipDataType HIP_R_32F,     // 内部计算精度（如 HIP_R_32F）
    //     HIPBLAS_GEMM_DEFAULT             // 算法选择
    // );

}

Status add_fp32(hipblasHandle_t handle, float alpha=1.0f, const float* x, float* y, int m, int n) {
    hipblasSaxpy(
        handle,   // cuBLAS句柄
        m * n,        // 元素总数
        &alpha,   // α的指针
        x,      // 输入x
        1,        // 连续访问（步长1）
        y,      // 输入/输出y
        1         // 连续访问
    );
}

Status add_int8(hipblasHandle_t handle, float alpha=1.0f, const int8_t* x, const int8_t* y, int m, int n);

Status add_int16(hipblasHandle_t handle, float alpha=1.0f, const int16_t* x, const int16_t* y, int m, int n);

Status add_int32(hipblasHandle_t handle, float alpha=1.0f, const int32_t* x, const int32_t* y, int m, int n);

} // namespace cuda
} // namespace chatty