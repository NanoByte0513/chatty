
#include <hip/hip_runtime.h>
// #include "add.cuh"

// namespace chatty {
// namespace cuda {

// Status add(cublasHandle_t handle, float alpha, const Tensor& x, float beta, const Tensor& y, Tensor& out) {
//     DType dtype = x.dtype();
//     Shape shape = x.shape();
//     if(dtype != y.dtype() || shape != y.shape()) {
//         // TODO: log
//         return StatusCode::CHATTY_STATUS_FAILURE;
//     }
//     switch(dtype) {
//     case DType::Float16:
//         return add_fp16(
//             handle,
//             alpha, static_cast<const __half*>(x.data()), 
//             beta, static_cast<const __half*>(y.data()),
//             const_cast<__half*>(static_cast<const __half*>(out.data())),
//             shape[0], shape[1]
//         );
//     case DType::Float32:
//         return add_fp32(
//             handle, 
//             alpha, x.dataAsCstFloat(), 
//             beta, y.dataAsCstFloat(),
//             out.dataAsFloat(),
//             shape[0], shape[1]
//         );
//     default:
//         return StatusCode::CHATTY_STATUS_FAILURE;
//     }
    
// }

// Status add_fp16(cublasHandle_t handle, float alpha, const __half* x, float beta, const __half* y, __half* out, int m, int n) {
//     __half alpha_h = __float2half(alpha);
//     __half beta_h = __float2half(beta);
//     cublasStatus_t status = cublasHgemm(handle,
//                                 CUBLAS_OP_N, CUBLAS_OP_N,
//                                 m, n, 1,       // k=1
//                                 &alpha_h,          // 直接使用__half*类型
//                                 x, lda,
//                                 y, ldb,
//                                 &beta_h,
//                                 out, ldc
//                             );

// }

// Status add_fp32(cublasHandle_t handle, float alpha, const float* x, float beta, const float* y, float* out, int m, int n) {
//     cublasStatus_t status = cublasSaxpy(
//                                 handle,   // cuBLAS句柄
//                                 m * n,        // 元素总数
//                                 &alpha,   // α的指针
//                                 x,      // 输入x
//                                 1,        // 连续访问（步长1）
//                                 y,      // 输入/输出y
//                                 1         // 连续访问
//                             );
//     if(status == CUBLAS_STATUS_SUCCESS) {
//         return StatusCode::CHATTY_STATUS_SUCCESS;
//     } else {
//         return StatusCode::CHATTY_STATUS_FAILURE;
//     }
// }

// Status add_int8(cublasHandle_t handle, float alpha=1.0f, const int8_t* x, const int8_t* y, int m, int n);

// Status add_int16(cublasHandle_t handle, float alpha=1.0f, const int16_t* x, const int16_t* y, int m, int n);

// Status add_int32(cublasHandle_t handle, float alpha=1.0f, const int32_t* x, const int32_t* y, int m, int n);

// } // namespace cuda
// } // namespace chatty