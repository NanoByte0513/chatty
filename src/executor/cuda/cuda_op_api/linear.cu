#include "hip/hip_runtime.h"
#include "linear.cuh"
#include "hipblas.h"

namespace chatty {
namespace cuda {
Status linear(hipblasHandle_t handle, const Tensor& x, const Tensor& weight, const Tensor& bias, Tensor& out) {
    DType input_dtype = x.dtype();
    DType weight_dtype = weight.dtype();
    int m = x.shape()[0];
    int k = x.shape()[1];
    int n = weight.shape()[1];
    switch(weight_dtype) {
    case DType::Float16:
        switch(input_dtype) {
        case DType::Float16:
            return linear_fp16(handle, (const __half*)x.data(), (const __half*)weight.data(), (const __half*)bias.data(), out.dataAsFloat(), m, k, n);
        }

    case DType::BF16:
        switch(input_dtype) {
        case DType::BF16:
            return linear_bf16(handle, (const __hip_bfloat16*)x.data(), (const __hip_bfloat16*)weight.data(), out.dataAsFloat(), m, k, n);
        }

    default:
        return StatusCode::CHATTY_STATUS_FAILURE;
    }
    return StatusCode::CHATTY_STATUS_SUCCESS;
}

Status linear(hipblasHandle_t handle, const Tensor& x, const Tensor& weight, Tensor& out) {
    DType input_dtype = x.dtype();
    DType weight_dtype = weight.dtype();
    int m = x.shape()[0];
    int k = x.shape()[1];
    int n = weight.shape()[1];
    switch(weight_dtype) {
    case DType::Float16:
        switch(input_dtype) {
        case DType::Float16:
            return linear_fp16(handle, (const __half*)x.data(), (const __half*)weight.data(), nullptr, out.dataAsFloat(), m, k, n);
        }
    default:
        return StatusCode::CHATTY_STATUS_FAILURE;
    }
    return StatusCode::CHATTY_STATUS_SUCCESS;
}

// Status linear_fp16(hipblasHandle_t handle, const __half* x, const __half* weight, const __half* bias, float* out, int m, int k, int n) {
//     // C = alpha * AB + beta * C
//     hipblasStatus_t status = hipblasGemmEx(
//         hipblasHandle_t handle,          // cuBLAS 上下文句柄
//         hipblasOperation_t transA,       // A 是否转置
//         hipblasOperation_t transB,       // B 是否转置
//         int m, int n, int k,            // 矩阵维度（C = m×n, A = m×k, B = k×n）
//         const void *alpha,              // 缩放因子（float 或 __half 指针）
//         const void *A, hipDataType Atype, int lda,  // A 矩阵及参数
//         const void *B, hipDataType Btype, int ldb,  // B 矩阵及参数
//         const void *beta,               // C 的缩放因子
//         void *C, hipDataType Ctype, int ldc, // C 矩阵及参数
//         hipDataType ComputeType,       // 计算精度（如 HIP_R_32F）
//         hipblasGemmAlgo_t algo           // 算法选择（启用 Tensor Core）
//     );
// }

Status linear_bf16(hipblasHandle_t handle, const __hip_bfloat16* x, const __hip_bfloat16* weight, float* out, int m, int k, int n) {
    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasGemmEx(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N, // A、B是否需要转置
        m, n, k,
        &alpha,
        weight, HIP_R_16BF, n,      // A类型为BF16
        x, HIP_R_16BF, k,      // B类型为BF16
        &beta,
        out, HIP_R_32F, n,       // C类型为FP32（累加防精度损失）
        HIP_R_32F,               // 计算精度为FP32
        CUBLAS_GEMM_DEFAULT_TENSOR_OP // 启用Tensor Core
    );
    return StatusCode::CHATTY_STATUS_SUCCESS;
}

} // namespace cuda
} // namespace chatty
