#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <vector>
#include <chrono>

#define CHECK_CUDA(func) \
    do { \
        hipError_t status = (func); \
        if (status != hipSuccess) { \
            std::cerr << "CUDA Error at " << __FILE__ << ":" << __LINE__ << ": " \
                      << hipGetErrorString(status) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#define CHECK_CUBLAS(func) \
    do { \
        hipblasStatus_t status = (func); \
        if (status != HIPBLAS_STATUS_SUCCESS) { \
            std::cerr << "cuBLAS Error at " << __FILE__ << ":" << __LINE__ << ": " \
                      << status << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

int main() {
    // 1. 初始化矩阵参数（可修改）
    const int M = 1024; // 矩阵A的行数
    const int N = 1024; // 矩阵B的列数
    const int K = 1024; // 矩阵A的列数/矩阵B的行数
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // 2. 创建主机端数据
    std::vector<float> h_A(M * K);
    std::vector<float> h_B(K * N);
    std::vector<float> h_C(M * N);
    std::vector<float> h_C_ref(M * N); // 用于验证的CPU结果

    // 初始化随机数据
    for (int i = 0; i < M * K; ++i) h_A[i] = static_cast<float>(rand()) / RAND_MAX;
    for (int i = 0; i < K * N; ++i) h_B[i] = static_cast<float>(rand()) / RAND_MAX;

    // 3. 分配设备端内存
    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, M * K * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_B, K * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_C, M * N * sizeof(float)));

    // 4. 初始化cuBLAS句柄
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // 5. 数据传输到GPU
    CHECK_CUBLAS(hipblasSetMatrix(M, K, sizeof(float), h_A.data(), M, d_A, M));
    CHECK_CUBLAS(hipblasSetMatrix(K, N, sizeof(float), h_B.data(), K, d_B, K));

    // 6. 执行矩阵乘法（核心操作）
    auto start = std::chrono::high_resolution_clock::now();
    CHECK_CUBLAS(
        hipblasSgemm(handle,
                    HIPBLAS_OP_N,  // A不转置（因cuBLAS默认列优先，此处实际需转置）
                    HIPBLAS_OP_N,  // B不转置
                    M,            // 结果矩阵行数
                    N,            // 结果矩阵列数
                    K,            // 累加维度
                    &alpha,
                    d_A, M,       // A的leading dimension = M
                    d_B, K,       // B的leading dimension = K
                    &beta,
                    d_C, M)       // C的leading dimension = M
    );
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    // 7. 取回结果
    CHECK_CUBLAS(hipblasGetMatrix(M, N, sizeof(float), d_C, M, h_C.data(), M));

    // 8. 计算耗时
    float duration_ms = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / 1000.0f;
    float gflops = (2.0f * M * N * K) / (duration_ms * 1e6); // 计算GFLOPS

    // 9. 验证结果（简单验证前10个元素）
    for (int i = 0; i < 10; ++i) {
        int row = rand() % M, col = rand() % N;
        float cpu_val = 0.0f;
        for (int k = 0; k < K; ++k) {
            cpu_val += h_A[row * K + k] * h_B[k * N + col];
        }
        float diff = fabs(h_C[row * N + col] - cpu_val);
        if (diff > 1e-5) {
            std::cerr << "Validation failed at (" << row << "," << col << "): "
                      << "GPU=" << h_C[row * N + col] << ", CPU=" << cpu_val << std::endl;
        }
    }

    // 10. 输出性能数据
    std::cout << "Matrix: [" << M << "x" << K << "] * [" << K << "x" << N << "]\n"
              << "Time: " << duration_ms << " ms\n"
              << "Performance: " << gflops << " GFLOPS" << std::endl;

    // 11. 清理资源
    CHECK_CUBLAS(hipblasDestroy(handle));
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));

    return 0;
}